#include "hip/hip_runtime.h"
#include "resnet.cuh"

#include <hiprand.h>
#include <hipblas.h>
#include <torch/torch.h>

static void resnet_init(INeuralNet* self, const IGame* game) {
    ResNetWrapper* wrapper = (ResNetWrapper*)self;
    ResNetModel* model = &wrapper->model;
    
    // Initialize model configuration based on game
    init_model_config(model, game);

    // Initialize cuDNN
    hipdnnCreate(&model->cudnn_handle);

    // Initialize input block
    init_input_block(model);

    // Initialize residual blocks
    init_residual_blocks(model);

    // Initialize output block
    init_output_block(model);

    // Initialize weights with small random values
    init_weights(model);
}

static void resnet_train(INeuralNet* self, TrainingExample* examples, int num_examples) {
    ResNetWrapper* wrapper = (ResNetWrapper*)self;
    ResNetModel* model = &wrapper->model;

    // Create cuBLAS handle
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    // Create Adam optimizer
    AdamOptimizer optimizer;
    init_adam_optimizer(&optimizer, model->learning_rate, model->weight_decay);

    // Training loop
    for (int epoch = 0; epoch < model->config.epochs; epoch++) {
        printf("EPOCH ::: %d\n", epoch + 1);

        float pi_loss_sum = 0.0f;
        float v_loss_sum = 0.0f;
        int batch_count = num_examples / model->config.batch_size;

        for (int batch = 0; batch < batch_count; batch++) {
            // Prepare batch data
            float* batch_boards;
            float* batch_pis;
            float* batch_vs;
            prepare_batch(examples, num_examples, model->config.batch_size, 
                          &batch_boards, &batch_pis, &batch_vs);

            // Forward pass
            float* out_pi;
            float* out_v;
            forward_resnet(model, batch_boards, &out_pi, &out_v);

            // Compute losses
            float l_pi = compute_policy_loss(batch_pis, out_pi, model->config.batch_size, model->config.action_size);
            float l_v = compute_value_loss(batch_vs, out_v, model->config.batch_size);
            float total_loss = l_pi + l_v;

            // Backward pass
            backward_resnet(model, batch_boards, batch_pis, batch_vs, out_pi, out_v);

            // Update weights
            adam_update(&optimizer, model);

            // Record loss
            pi_loss_sum += l_pi;
            v_loss_sum += l_v;

            // Clean up
            hipFree(batch_boards);
            hipFree(batch_pis);
            hipFree(batch_vs);
            hipFree(out_pi);
            hipFree(out_v);
        }

        // Print epoch results
        printf("Average Policy Loss: %f, Average Value Loss: %f\n", 
               pi_loss_sum / batch_count, v_loss_sum / batch_count);
    }

    // Clean up
    hipblasDestroy(cublas_handle);
}

static void resnet_predict(INeuralNet* self, const float* board, float* pi, float* v) {
    ResNetWrapper* wrapper = (ResNetWrapper*)self;
    ResNetModel* model = &wrapper->model;

    // Allocate device memory for input and output
    float *d_board, *d_pi, *d_v;
    hipMalloc(&d_board, sizeof(float) * model->config.input_channels * model->config.input_height * model->config.input_width);
    hipMalloc(&d_pi, sizeof(float) * model->config.num_actions);
    hipMalloc(&d_v, sizeof(float));

    // Copy input to device
    hipMemcpy(d_board, board, sizeof(float) * model->config.input_channels * model->config.input_height * model->config.input_width, hipMemcpyHostToDevice);

    // Forward pass
    forward_resnet(model, d_board, &d_pi, &d_v);

    // Copy output back to host
    hipMemcpy(pi, d_pi, sizeof(float) * model->config.num_actions, hipMemcpyDeviceToHost);
    hipMemcpy(v, d_v, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_board);
    hipFree(d_pi);
    hipFree(d_v);
}

static void resnet_save_checkpoint(INeuralNet* self, const char* folder, const char* filename) {
    ResNetWrapper* wrapper = (ResNetWrapper*)self;
    ResNetModel* model = &wrapper->model;

    char filepath[MAX_FILENAME_LENGTH];
    snprintf(filepath, MAX_FILENAME_LENGTH, "%s/%s", folder, filename);

    FILE* file = fopen(filepath, "wb");
    if (file == NULL) {
        fprintf(stderr, "Error: Unable to open file for writing: %s\n", filepath);
        return;
    }

    // Save model configuration
    fwrite(&model->config, sizeof(ModelConfig), 1, file);

    // Save weights
    // Input block
    fwrite(model->input_conv_weights, sizeof(float), model->config.num_filters * model->config.input_channels * 3 * 3, file);
    fwrite(model->input_bn_scale, sizeof(float), model->config.num_filters, file);
    fwrite(model->input_bn_bias, sizeof(float), model->config.num_filters, file);

    // Residual blocks
    for (int i = 0; i < model->config.num_residual_blocks * 2; i++) {
        fwrite(model->res_conv_weights[i], sizeof(float), model->config.num_filters * model->config.num_filters * 3 * 3, file);
        fwrite(model->res_bn_scales[i], sizeof(float), model->config.num_filters, file);
        fwrite(model->res_bn_biases[i], sizeof(float), model->config.num_filters, file);
    }

    // Output block
    fwrite(model->value_conv_weights, sizeof(float), model->config.num_filters, file);
    fwrite(model->value_bn_scale, sizeof(float), 1, file);
    fwrite(model->value_bn_bias, sizeof(float), 1, file);
    fwrite(model->value_fc1_weights, sizeof(float), model->config.input_height * model->config.input_width * 256, file);
    fwrite(model->value_fc1_bias, sizeof(float), 256, file);
    fwrite(model->value_fc2_weights, sizeof(float), 256, file);
    fwrite(model->value_fc2_bias, sizeof(float), 1, file);

    fwrite(model->policy_conv_weights, sizeof(float), 2 * model->config.num_filters, file);
    fwrite(model->policy_bn_scale, sizeof(float), 2, file);
    fwrite(model->policy_bn_bias, sizeof(float), 2, file);
    fwrite(model->policy_fc_weights, sizeof(float), 2 * model->config.input_height * model->config.input_width * model->config.num_actions, file);
    fwrite(model->policy_fc_bias, sizeof(float), model->config.num_actions, file);

    fclose(file);
}

static void resnet_load_checkpoint(INeuralNet* self, const char* folder, const char* filename) {
    ResNetWrapper* wrapper = (ResNetWrapper*)self;
    ResNetModel* model = &wrapper->model;

    char filepath[MAX_FILENAME_LENGTH];
    snprintf(filepath, MAX_FILENAME_LENGTH, "%s/%s", folder, filename);

    FILE* file = fopen(filepath, "rb");
    if (file == NULL) {
        fprintf(stderr, "Error: Unable to open file for reading: %s\n", filepath);
        return;
    }

    // Load model configuration
    fread(&model->config, sizeof(ModelConfig), 1, file);

    // Load weights
    // Input block
    hipMemcpy(model->input_conv_weights, model->input_conv_weights, sizeof(float) * model->config.num_filters * model->config.input_channels * 3 * 3, hipMemcpyHostToDevice);
    hipMemcpy(model->input_bn_scale, model->input_bn_scale, sizeof(float) * model->config.num_filters, hipMemcpyHostToDevice);
    hipMemcpy(model->input_bn_bias, model->input_bn_bias, sizeof(float) * model->config.num_filters, hipMemcpyHostToDevice);

    // Residual blocks
    for (int i = 0; i < model->config.num_residual_blocks * 2; i++) {
        hipMemcpy(model->res_conv_weights[i], model->res_conv_weights[i], sizeof(float) * model->config.num_filters * model->config.num_filters * 3 * 3, hipMemcpyHostToDevice);
        hipMemcpy(model->res_bn_scales[i], model->res_bn_scales[i], sizeof(float) * model->config.num_filters, hipMemcpyHostToDevice);
        hipMemcpy(model->res_bn_biases[i], model->res_bn_biases[i], sizeof(float) * model->config.num_filters, hipMemcpyHostToDevice);
    }

    // Output block
    hipMemcpy(model->value_conv_weights, model->value_conv_weights, sizeof(float) * model->config.num_filters, hipMemcpyHostToDevice);
    hipMemcpy(model->value_bn_scale, model->value_bn_scale, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(model->value_bn_bias, model->value_bn_bias, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(model->value_fc1_weights, model->value_fc1_weights, sizeof(float) * model->config.input_height * model->config.input_width * 256, hipMemcpyHostToDevice);
    hipMemcpy(model->value_fc1_bias, model->value_fc1_bias, sizeof(float) * 256, hipMemcpyHostToDevice);
    hipMemcpy(model->value_fc2_weights, model->value_fc2_weights, sizeof(float) * 256, hipMemcpyHostToDevice);
    hipMemcpy(model->value_fc2_bias, model->value_fc2_bias, sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(model->policy_conv_weights, model->policy_conv_weights, sizeof(float) * 2 * model->config.num_filters, hipMemcpyHostToDevice);
    hipMemcpy(model->policy_bn_scale, model->policy_bn_scale, sizeof(float) * 2, hipMemcpyHostToDevice);
    hipMemcpy(model->policy_bn_bias, model->policy_bn_bias, sizeof(float) * 2, hipMemcpyHostToDevice);
    hipMemcpy(model->policy_fc_weights, model->policy_fc_weights, sizeof(float) * 2 * model->config.input_height * model->config.input_width * model->config.num_actions, hipMemcpyHostToDevice);
    hipMemcpy(model->policy_fc_bias, model->policy_fc_bias, sizeof(float) * model->config.num_actions, hipMemcpyHostToDevice);

    fclose(file);
}

static void resnet_destroy(INeuralNet* self) {
    ResNetWrapper* wrapper = (ResNetWrapper*)self;
    ResNetModel* model = &wrapper->model;

    // Free device memory
    hipFree(model->input_conv_weights);
    hipFree(model->input_bn_scale);
    hipFree(model->input_bn_bias);

    for (int i = 0; i < model->config.num_residual_blocks * 2; i++) {
        hipFree(model->res_conv_weights[i]);
        hipFree(model->res_bn_scales[i]);
        hipFree(model->res_bn_biases[i]);
    }

    hipFree(model->value_conv_weights);
    hipFree(model->value_bn_scale);
    hipFree(model->value_bn_bias);
    hipFree(model->value_fc1_weights);
    hipFree(model->value_fc1_bias);
    hipFree(model->value_fc2_weights);
    hipFree(model->value_fc2_bias);

    hipFree(model->policy_conv_weights);
    hipFree(model->policy_bn_scale);
    hipFree(model->policy_bn_bias);
    hipFree(model->policy_fc_weights);
    hipFree(model->policy_fc_bias);

    // Free host memory
    free(model->res_conv_filters);
    free(model->res_bn_means);
    free(model->res_bn_vars);
    free(model->res_conv_weights);
    free(model->res_bn_scales);
    free(model->res_bn_biases);

    // Destroy cuDNN handles
    hipdnnDestroy(model->cudnn_handle);

    // Free the wrapper
    free(wrapper);
}

INeuralNet* create_resnet_model(const IGame* game) {
    ResNetWrapper* wrapper = (ResNetWrapper*)malloc(sizeof(ResNetWrapper));
    wrapper->base.impl = wrapper;
    wrapper->base.init = resnet_init;
    wrapper->base.train = resnet_train;
    wrapper->base.predict = resnet_predict;
    wrapper->base.save_checkpoint = resnet_save_checkpoint;
    wrapper->base.load_checkpoint = resnet_load_checkpoint;
    wrapper->base.destroy = resnet_destroy;

    resnet_init(&wrapper->base, game);

    return &wrapper->base;
}
/*************************************************************************************************************************************************************
 * INIT HELPER FUNCTIONS
**************************************************************************************************************************************************************/

static void init_model_config(ResNetModel* model, const IGame* game) {
    // Set up model configuration based on game parameters
    int rows, cols;
    game->get_board_size(game, &rows, &cols);
    model->config.input_channels = 3;  // Assuming 3 channels for player 1, player 2, and turn
    model->config.input_height = rows;
    model->config.input_width = cols;
    model->config.num_actions = game->get_action_size(game);
    model->config.num_residual_blocks = 19;  // AlphaZero used 19 residual blocks
    model->config.num_filters = 256;
    model->config.learning_rate = 0.001;
    model->config.weight_decay = 0.0001;
}

static void init_input_block(ResNetModel* model) {
    // Create and initialize convolution filter descriptor
    hipdnnCreateFilterDescriptor(&model->input_conv_filter);
    hipdnnSetFilter4dDescriptor(model->input_conv_filter, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                               model->config.num_filters, model->config.input_channels,
                               3, 3);  // 3x3 convolution

    // Allocate memory for convolution weights
    hipMalloc(&model->input_conv_weights, sizeof(float) * model->config.num_filters * model->config.input_channels * 3 * 3);

    // Create and initialize batch normalization descriptors
    hipdnnCreateTensorDescriptor(&model->input_bn_mean);
    hipdnnCreateTensorDescriptor(&model->input_bn_var);
    hipdnnSetTensor4dDescriptor(model->input_bn_mean, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               1, model->config.num_filters, 1, 1);
    hipdnnSetTensor4dDescriptor(model->input_bn_var, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               1, model->config.num_filters, 1, 1);

    // Allocate memory for batch normalization parameters
    hipMalloc(&model->input_bn_scale, sizeof(float) * model->config.num_filters);
    hipMalloc(&model->input_bn_bias, sizeof(float) * model->config.num_filters);
}

static void init_residual_blocks(ResNetModel* model) {
    // Allocate arrays for residual block parameters
    model->res_conv_filters = malloc(sizeof(hipdnnFilterDescriptor_t) * model->config.num_residual_blocks * 2);
    model->res_bn_means = malloc(sizeof(hipdnnTensorDescriptor_t) * model->config.num_residual_blocks * 2);
    model->res_bn_vars = malloc(sizeof(hipdnnTensorDescriptor_t) * model->config.num_residual_blocks * 2);
    model->res_conv_weights = malloc(sizeof(float*) * model->config.num_residual_blocks * 2);
    model->res_bn_scales = malloc(sizeof(float*) * model->config.num_residual_blocks * 2);
    model->res_bn_biases = malloc(sizeof(float*) * model->config.num_residual_blocks * 2);

    for (int i = 0; i < model->config.num_residual_blocks; i++) {
        for (int j = 0; j < 2; j++) {
            int idx = i * 2 + j;
            // Create and initialize convolution filter descriptor
            hipdnnCreateFilterDescriptor(&model->res_conv_filters[idx]);
            hipdnnSetFilter4dDescriptor(model->res_conv_filters[idx], HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                                       model->config.num_filters, model->config.num_filters,
                                       3, 3);  // 3x3 convolution

            // Allocate memory for convolution weights
            hipMalloc(&model->res_conv_weights[idx], sizeof(float) * model->config.num_filters * model->config.num_filters * 3 * 3);

            // Create and initialize batch normalization descriptors
            hipdnnCreateTensorDescriptor(&model->res_bn_means[idx]);
            hipdnnCreateTensorDescriptor(&model->res_bn_vars[idx]);
            hipdnnSetTensor4dDescriptor(model->res_bn_means[idx], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                       1, model->config.num_filters, 1, 1);
            hipdnnSetTensor4dDescriptor(model->res_bn_vars[idx], HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                                       1, model->config.num_filters, 1, 1);

            // Allocate memory for batch normalization parameters
            hipMalloc(&model->res_bn_scales[idx], sizeof(float) * model->config.num_filters);
            hipMalloc(&model->res_bn_biases[idx], sizeof(float) * model->config.num_filters);
        }
    }
}

static void init_output_block(ResNetModel* model) {
    // Initialize value head
    hipdnnCreateFilterDescriptor(&model->value_conv_filter);
    hipdnnSetFilter4dDescriptor(model->value_conv_filter, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                               1, model->config.num_filters, 1, 1);  // 1x1 convolution
    hipMalloc(&model->value_conv_weights, sizeof(float) * model->config.num_filters);

    hipdnnCreateTensorDescriptor(&model->value_bn_mean);
    hipdnnCreateTensorDescriptor(&model->value_bn_var);
    hipdnnSetTensor4dDescriptor(model->value_bn_mean, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               1, 1, 1, 1);
    hipdnnSetTensor4dDescriptor(model->value_bn_var, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               1, 1, 1, 1);

    hipMalloc(&model->value_bn_scale, sizeof(float));
    hipMalloc(&model->value_bn_bias, sizeof(float));

    // Allocate memory for fully connected layers in value head
    int fc1_size = model->config.input_height * model->config.input_width;
    hipMalloc(&model->value_fc1_weights, sizeof(float) * fc1_size * 256);
    hipMalloc(&model->value_fc1_bias, sizeof(float) * 256);
    hipMalloc(&model->value_fc2_weights, sizeof(float) * 256);
    hipMalloc(&model->value_fc2_bias, sizeof(float));

    // Initialize policy head
    hipdnnCreateFilterDescriptor(&model->policy_conv_filter);
    hipdnnSetFilter4dDescriptor(model->policy_conv_filter, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
                               2, model->config.num_filters, 1, 1);  // 1x1 convolution
    hipMalloc(&model->policy_conv_weights, sizeof(float) * 2 * model->config.num_filters);

    hipdnnCreateTensorDescriptor(&model->policy_bn_mean);
    hipdnnCreateTensorDescriptor(&model->policy_bn_var);
    hipdnnSetTensor4dDescriptor(model->policy_bn_mean, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               1, 2, 1, 1);
    hipdnnSetTensor4dDescriptor(model->policy_bn_var, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               1, 2, 1, 1);

    hipMalloc(&model->policy_bn_scale, sizeof(float) * 2);
    hipMalloc(&model->policy_bn_bias, sizeof(float) * 2);

    // Allocate memory for fully connected layer in policy head
    int policy_fc_size = 2 * model->config.input_height * model->config.input_width;
    hipMalloc(&model->policy_fc_weights, sizeof(float) * policy_fc_size * model->config.num_actions);
    hipMalloc(&model->policy_fc_bias, sizeof(float) * model->config.num_actions);
}

static void init_weights(ResNetModel* model) {
    // Initialize weights with small random values
    // You can use cuRAND for this purpose
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    // Initialize input block weights
    hiprandGenerateNormal(gen, model->input_conv_weights, model->config.num_filters * model->config.input_channels * 3 * 3, 0, 0.1);
    hiprandGenerateNormal(gen, model->input_bn_scale, model->config.num_filters, 1, 0.1);
    hiprandGenerateNormal(gen, model->input_bn_bias, model->config.num_filters, 0, 0.1);

    // Initialize residual block weights
    for (int i = 0; i < model->config.num_residual_blocks * 2; i++) {
        hiprandGenerateNormal(gen, model->res_conv_weights[i], model->config.num_filters * model->config.num_filters * 3 * 3, 0, 0.1);
        hiprandGenerateNormal(gen, model->res_bn_scales[i], model->config.num_filters, 1, 0.1);
        hiprandGenerateNormal(gen, model->res_bn_biases[i], model->config.num_filters, 0, 0.1);
    }

    // Initialize output block weights
    hiprandGenerateNormal(gen, model->value_conv_weights, model->config.num_filters, 0, 0.1);
    hiprandGenerateNormal(gen, model->value_bn_scale, 1, 1, 0.1);
    hiprandGenerateNormal(gen, model->value_bn_bias, 1, 0, 0.1);
    hiprandGenerateNormal(gen, model->value_fc1_weights, model->config.input_height * model->config.input_width * 256, 0, 0.1);
    hiprandGenerateNormal(gen, model->value_fc1_bias, 256, 0, 0.1);
    hiprandGenerateNormal(gen, model->value_fc2_weights, 256, 0, 0.1);
    hiprandGenerateNormal(gen, model->value_fc2_bias, 1, 0, 0.1);

    hiprandGenerateNormal(gen, model->policy_conv_weights, 2 * model->config.num_filters, 0, 0.1);
    hiprandGenerateNormal(gen, model->policy_bn_scale, 2, 1, 0.1);
    hiprandGenerateNormal(gen, model->policy_bn_bias, 2, 0, 0.1);
    hiprandGenerateNormal(gen, model->policy_fc_weights, 2 * model->config.input_height * model->config.input_width * model->config.num_actions, 0, 0.1);
    hiprandGenerateNormal(gen, model->policy_fc_bias, model->config.num_actions, 0, 0.1);

    hiprandDestroyGenerator(gen);
}

/*************************************************************************************************************************************************************
 * TRAIN HELPER FUNCTIONS
**************************************************************************************************************************************************************/

torch::optim::Adam init_adam_optimizer(ResNetModel* model, float learning_rate, float weight_decay) {
    std::vector<torch::Tensor> params;
    
    // Input block
    params.push_back(torch::from_blob(model->input_conv_weights, {model->config.num_filters, model->config.input_channels, 3, 3}, torch::kCUDA));
    params.push_back(torch::from_blob(model->input_bn_scale, {model->config.num_filters}, torch::kCUDA));
    params.push_back(torch::from_blob(model->input_bn_bias, {model->config.num_filters}, torch::kCUDA));

    // Residual blocks
    for (int i = 0; i < model->config.num_residual_blocks * 2; i++) {
        params.push_back(torch::from_blob(model->res_conv_weights[i], {model->config.num_filters, model->config.num_filters, 3, 3}, torch::kCUDA));
        params.push_back(torch::from_blob(model->res_bn_scales[i], {model->config.num_filters}, torch::kCUDA));
        params.push_back(torch::from_blob(model->res_bn_biases[i], {model->config.num_filters}, torch::kCUDA));
    }

    // Value head
    params.push_back(torch::from_blob(model->value_conv_weights, {1, model->config.num_filters, 1, 1}, torch::kCUDA));
    params.push_back(torch::from_blob(model->value_bn_scale, {1}, torch::kCUDA));
    params.push_back(torch::from_blob(model->value_bn_bias, {1}, torch::kCUDA));
    params.push_back(torch::from_blob(model->value_fc1_weights, {256, model->config.input_height * model->config.input_width}, torch::kCUDA));
    params.push_back(torch::from_blob(model->value_fc1_bias, {256}, torch::kCUDA));
    params.push_back(torch::from_blob(model->value_fc2_weights, {1, 256}, torch::kCUDA));
    params.push_back(torch::from_blob(model->value_fc2_bias, {1}, torch::kCUDA));

    // Policy head
    params.push_back(torch::from_blob(model->policy_conv_weights, {2, model->config.num_filters, 1, 1}, torch::kCUDA));
    params.push_back(torch::from_blob(model->policy_bn_scale, {2}, torch::kCUDA));
    params.push_back(torch::from_blob(model->policy_bn_bias, {2}, torch::kCUDA));
    params.push_back(torch::from_blob(model->policy_fc_weights, {model->config.num_actions, 2 * model->config.input_height * model->config.input_width}, torch::kCUDA));
    params.push_back(torch::from_blob(model->policy_fc_bias, {model->config.num_actions}, torch::kCUDA));

    return torch::optim::Adam(params, torch::optim::AdamOptions(learning_rate).weight_decay(weight_decay));
}

void prepare_batch(TrainingExample* examples, int num_examples, int batch_size,
                   float** batch_boards, float** batch_pis, float** batch_vs) {
    // Allocate memory for batch data
    hipMalloc(batch_boards, batch_size * BOARD_SIZE * sizeof(float));
    hipMalloc(batch_pis, batch_size * ACTION_SIZE * sizeof(float));
    hipMalloc(batch_vs, batch_size * sizeof(float));

    // Use cuRAND for random selection
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    int* d_indices;
    hipMalloc(&d_indices, batch_size * sizeof(int));
    hiprandGenerate(gen, (unsigned int*)d_indices, batch_size);

    // Custom CUDA kernel to prepare batch
    prepare_batch_kernel<<<(batch_size + 255) / 256, 256>>>(
        examples, num_examples, d_indices, *batch_boards, *batch_pis, *batch_vs, batch_size);

    hipFree(d_indices);
    hiprandDestroyGenerator(gen);
}

__global__ void prepare_batch_kernel(TrainingExample* examples, int num_examples, int* indices,
                                     float* batch_boards, float* batch_pis, float* batch_vs, int batch_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size) {
        int example_idx = indices[idx] % num_examples;
        memcpy(batch_boards + idx * BOARD_SIZE, examples[example_idx].board, BOARD_SIZE * sizeof(float));
        memcpy(batch_pis + idx * ACTION_SIZE, examples[example_idx].pi, ACTION_SIZE * sizeof(float));
        batch_vs[idx] = examples[example_idx].v;
    }
}

void forward_resnet(ResNetModel* model, float* batch_boards, float** out_pi, float** out_v) {
    hipdnnHandle_t cudnn = model->cudnn_handle;
    float alpha = 1.0f, beta = 0.0f;
    
    hipdnnTensorDescriptor_t input_descriptor, output_descriptor;
    hipdnnFilterDescriptor_t filter_descriptor;
    hipdnnConvolutionDescriptor_t conv_descriptor;
    hipdnnActivationDescriptor_t activation_descriptor;
    hipdnnTensorDescriptor_t bn_descriptor;
    
    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnCreateFilterDescriptor(&filter_descriptor);
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);
    hipdnnCreateActivationDescriptor(&activation_descriptor);
    hipdnnCreateTensorDescriptor(&bn_descriptor);
    
    hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               model->config.batch_size, model->config.input_channels,
                               model->config.input_height, model->config.input_width);
    
    hipdnnSetActivationDescriptor(activation_descriptor, HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0.0);
    
    float* output;
    hipMalloc(&output, model->config.batch_size * model->config.num_filters * 
               model->config.input_height * model->config.input_width * sizeof(float));
    
    // Input convolution
    hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, batch_boards,
                            model->input_conv_filter, model->input_conv_weights,
                            conv_descriptor, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                            model->workspace, model->workspace_size, &beta, output_descriptor, output);
    
    // Batch normalization
    hipdnnBatchNormalizationForwardTraining(cudnn, HIPDNN_BATCHNORM_SPATIAL,
                                           &alpha, &beta, input_descriptor, output,
                                           output_descriptor, output,
                                           bn_descriptor, model->input_bn_scale, model->input_bn_bias,
                                           1.0, model->input_bn_mean, model->input_bn_var,
                                           HIPDNN_BN_MIN_EPSILON, model->input_bn_mean, model->input_bn_var);
    
    // ReLU activation
    hipdnnActivationForward(cudnn, activation_descriptor, &alpha, input_descriptor, output,
                           &beta, output_descriptor, output);
    
    // Residual blocks
    float* prev_output = output;
    for (int i = 0; i < model->config.num_residual_blocks; i++) {
        float* res_output;
        hipMalloc(&res_output, model->config.batch_size * model->config.num_filters * 
                   model->config.input_height * model->config.input_width * sizeof(float));
        
        // First convolution in residual block
        hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, prev_output,
                                model->res_conv_filters[i*2], model->res_conv_weights[i*2],
                                conv_descriptor, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                model->workspace, model->workspace_size, &beta, output_descriptor, res_output);
        
        hipdnnBatchNormalizationForwardTraining(cudnn, HIPDNN_BATCHNORM_SPATIAL,
                                               &alpha, &beta, input_descriptor, res_output,
                                               output_descriptor, res_output,
                                               bn_descriptor, model->res_bn_scales[i*2], model->res_bn_biases[i*2],
                                               1.0, model->res_bn_means[i*2], model->res_bn_vars[i*2],
                                               HIPDNN_BN_MIN_EPSILON, model->res_bn_means[i*2], model->res_bn_vars[i*2]);
        
        hipdnnActivationForward(cudnn, activation_descriptor, &alpha, input_descriptor, res_output,
                               &beta, output_descriptor, res_output);
        
        // Second convolution in residual block
        hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, res_output,
                                model->res_conv_filters[i*2+1], model->res_conv_weights[i*2+1],
                                conv_descriptor, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                model->workspace, model->workspace_size, &beta, output_descriptor, res_output);
        
        hipdnnBatchNormalizationForwardTraining(cudnn, HIPDNN_BATCHNORM_SPATIAL,
                                               &alpha, &beta, input_descriptor, res_output,
                                               output_descriptor, res_output,
                                               bn_descriptor, model->res_bn_scales[i*2+1], model->res_bn_biases[i*2+1],
                                               1.0, model->res_bn_means[i*2+1], model->res_bn_vars[i*2+1],
                                               HIPDNN_BN_MIN_EPSILON, model->res_bn_means[i*2+1], model->res_bn_vars[i*2+1]);
        
        // Add residual connection
        hipMemcpy(output, prev_output, model->config.batch_size * model->config.num_filters * 
                   model->config.input_height * model->config.input_width * sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(output, res_output, model->config.batch_size * model->config.num_filters * 
                   model->config.input_height * model->config.input_width * sizeof(float), hipMemcpyDeviceToDevice);
        
        hipdnnActivationForward(cudnn, activation_descriptor, &alpha, input_descriptor, output,
                               &beta, output_descriptor, output);
        
        hipFree(res_output);
        prev_output = output;
    }
    
    // Policy head
    float* policy_output;
    hipMalloc(&policy_output, model->config.batch_size * 2 * model->config.input_height * model->config.input_width * sizeof(float));
    
    hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, output,
                            model->policy_conv_filter, model->policy_conv_weights,
                            conv_descriptor, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                            model->workspace, model->workspace_size, &beta, output_descriptor, policy_output);
    
    hipdnnBatchNormalizationForwardTraining(cudnn, HIPDNN_BATCHNORM_SPATIAL,
                                           &alpha, &beta, input_descriptor, policy_output,
                                           output_descriptor, policy_output,
                                           bn_descriptor, model->policy_bn_scale, model->policy_bn_bias,
                                           1.0, model->policy_bn_mean, model->policy_bn_var,
                                           HIPDNN_BN_MIN_EPSILON, model->policy_bn_mean, model->policy_bn_var);
    
    // Use PyTorch for the fully connected layer and softmax
    auto policy_tensor = torch::from_blob(policy_output, {model->config.batch_size, 2, model->config.input_height, model->config.input_width}, torch::kCUDA);
    auto policy_fc_weight = torch::from_blob(model->policy_fc_weights, {model->config.num_actions, 2 * model->config.input_height * model->config.input_width}, torch::kCUDA);
    auto policy_fc_bias = torch::from_blob(model->policy_fc_bias, {model->config.num_actions}, torch::kCUDA);
    
    policy_tensor = torch::nn::functional::linear(policy_tensor.view({model->config.batch_size, -1}), policy_fc_weight, policy_fc_bias);
    policy_tensor = torch::nn::functional::softmax(policy_tensor, /*dim=*/1);
    
    // Value head
    float* value_output;
    hipMalloc(&value_output, model->config.batch_size * model->config.input_height * model->config.input_width * sizeof(float));
    
    hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, output,
                            model->value_conv_filter, model->value_conv_weights,
                            conv_descriptor, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                            model->workspace, model->workspace_size, &beta, output_descriptor, value_output);
    
    hipdnnBatchNormalizationForwardTraining(cudnn, HIPDNN_BATCHNORM_SPATIAL,
                                           &alpha, &beta, input_descriptor, value_output,
                                           output_descriptor, value_output,
                                           bn_descriptor, model->value_bn_scale, model->value_bn_bias,
                                           1.0, model->value_bn_mean, model->value_bn_var,
                                           HIPDNN_BN_MIN_EPSILON, model->value_bn_mean, model->value_bn_var);
    
    auto value_tensor = torch::from_blob(value_output, {model->config.batch_size, 1, model->config.input_height, model->config.input_width}, torch::kCUDA);
    auto value_fc1_weight = torch::from_blob(model->value_fc1_weights, {256, model->config.input_height * model->config.input_width}, torch::kCUDA);
    auto value_fc1_bias = torch::from_blob(model->value_fc1_bias, {256}, torch::kCUDA);
    auto value_fc2_weight = torch::from_blob(model->value_fc2_weights, {1, 256}, torch::kCUDA);
    auto value_fc2_bias = torch::from_blob(model->value_fc2_bias, {1}, torch::kCUDA);
    
    value_tensor = torch::nn::functional::linear(value_tensor.view({model->config.batch_size, -1}), value_fc1_weight, value_fc1_bias);
    value_tensor = torch::relu(value_tensor);
    value_tensor = torch::nn::functional::linear(value_tensor, value_fc2_weight, value_fc2_bias);
    value_tensor = torch::tanh(value_tensor);
    
    // Copy results back to output pointers
    hipMalloc(out_pi, model->config.batch_size * model->config.num_actions * sizeof(float));
    hipMalloc(out_v, model->config.batch_size * sizeof(float));
    hipMemcpy(*out_pi, policy_tensor.data_ptr(), model->config.batch_size * model->config.num_actions * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(*out_v, value_tensor.data_ptr(), model->config.batch_size * sizeof(float), hipMemcpyDeviceToDevice);
    
    // Cleanup
    hipFree(output);
    hipFree(policy_output);
    hipFree(value_output);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);
    hipdnnDestroyActivationDescriptor(activation_descriptor);
    hipdnnDestroyTensorDescriptor(bn_descriptor);
}

__global__ void policy_loss_kernel(float* target_pi, float* out_pi, float* loss, int batch_size, int action_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size) {
        float sample_loss = 0.0f;
        for (int a = 0; a < action_size; a++) {
            int i = idx * action_size + a;
            sample_loss -= target_pi[i] * logf(out_pi[i] + 1e-8f);
        }
        loss[idx] = sample_loss;
    }
}

__global__ void value_loss_kernel(float* target_v, float* out_v, float* loss, int batch_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size) {
        float diff = target_v[idx] - out_v[idx];
        loss[idx] = diff * diff;
    }
}

std::pair<float, float> compute_losses(float* target_pi, float* target_v, float* out_pi, float* out_v, int batch_size, int action_size) {
    auto target_pi_tensor = torch::from_blob(target_pi, {batch_size, action_size}, torch::kCUDA);
    auto target_v_tensor = torch::from_blob(target_v, {batch_size}, torch::kCUDA);
    auto out_pi_tensor = torch::from_blob(out_pi, {batch_size, action_size}, torch::kCUDA);
    auto out_v_tensor = torch::from_blob(out_v, {batch_size}, torch::kCUDA);

    auto pi_loss = torch::nn::functional::kl_div(out_pi_tensor.log(), target_pi_tensor, torch::kSum);
    auto v_loss = torch::mse_loss(out_v_tensor, target_v_tensor, torch::kSum);

    return {pi_loss.item<float>() / batch_size, v_loss.item<float>() / batch_size};
}

void backward_resnet(ResNetModel* model, float* batch_boards, float* target_pi, float* target_v, float* out_pi, float* out_v) {
    hipdnnHandle_t cudnn = model->cudnn_handle;
    float alpha = 1.0f, beta = 0.0f;

    // Convert inputs to PyTorch tensors
    auto boards_tensor = torch::from_blob(batch_boards, {model->config.batch_size, model->config.input_channels, model->config.input_height, model->config.input_width}, torch::kCUDA).requires_grad_();
    auto target_pi_tensor = torch::from_blob(target_pi, {model->config.batch_size, model->config.num_actions}, torch::kCUDA);
    auto target_v_tensor = torch::from_blob(target_v, {model->config.batch_size}, torch::kCUDA);
    auto out_pi_tensor = torch::from_blob(out_pi, {model->config.batch_size, model->config.num_actions}, torch::kCUDA).requires_grad_();
    auto out_v_tensor = torch::from_blob(out_v, {model->config.batch_size}, torch::kCUDA).requires_grad_();

    // Compute losses
    auto pi_loss = torch::nn::functional::kl_div(out_pi_tensor.log(), target_pi_tensor);
    auto v_loss = torch::mse_loss(out_v_tensor, target_v_tensor);
    auto total_loss = pi_loss + v_loss;

    // Backward pass
    total_loss.backward();

    // Get gradients
    auto d_out_pi = out_pi_tensor.grad().contiguous();
    auto d_out_v = out_v_tensor.grad().contiguous();

    hipdnnTensorDescriptor_t input_descriptor, output_descriptor;
    hipdnnFilterDescriptor_t filter_descriptor;
    hipdnnConvolutionDescriptor_t conv_descriptor;

    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnCreateFilterDescriptor(&filter_descriptor);
    hipdnnCreateConvolutionDescriptor(&conv_descriptor);

    hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
                               model->config.batch_size, model->config.num_filters,
                               model->config.input_height, model->config.input_width);

    // Policy head backward
    float* d_policy_conv;
    hipMalloc(&d_policy_conv, model->config.batch_size * 2 * model->config.input_height * model->config.input_width * sizeof(float));

    // FC layer backward (using PyTorch)
    auto d_policy_fc = torch::nn::functional::linear(d_out_pi, 
                                                     torch::from_blob(model->policy_fc_weights, {model->config.num_actions, 2 * model->config.input_height * model->config.input_width}, torch::kCUDA).t());
    hipMemcpy(d_policy_conv, d_policy_fc.data_ptr(), model->config.batch_size * 2 * model->config.input_height * model->config.input_width * sizeof(float), hipMemcpyDeviceToDevice);

    // Policy convolution backward
    hipdnnConvolutionBackwardData(cudnn, &alpha, model->policy_conv_filter, model->policy_conv_weights,
                                 output_descriptor, d_policy_conv, conv_descriptor,
                                 HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, model->workspace, model->workspace_size,
                                 &beta, input_descriptor, model->d_policy_conv_weights);

    hipdnnConvolutionBackwardFilter(cudnn, &alpha, input_descriptor, batch_boards,
                                   output_descriptor, d_policy_conv, conv_descriptor,
                                   HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0, model->workspace, model->workspace_size,
                                   &beta, model->policy_conv_filter, model->d_policy_conv_weights);

    // Value head backward
    float* d_value_conv;
    hipMalloc(&d_value_conv, model->config.batch_size * model->config.input_height * model->config.input_width * sizeof(float));

    // FC layers backward (using PyTorch)
    auto d_value_fc2 = torch::nn::functional::linear(d_out_v, 
                                                     torch::from_blob(model->value_fc2_weights, {1, 256}, torch::kCUDA).t());
    auto d_value_fc1 = torch::nn::functional::linear(torch::relu(d_value_fc2), 
                                                     torch::from_blob(model->value_fc1_weights, {256, model->config.input_height * model->config.input_width}, torch::kCUDA).t());
    hipMemcpy(d_value_conv, d_value_fc1.data_ptr(), model->config.batch_size * model->config.input_height * model->config.input_width * sizeof(float), hipMemcpyDeviceToDevice);

    // Value convolution backward
    hipdnnConvolutionBackwardData(cudnn, &alpha, model->value_conv_filter, model->value_conv_weights,
                                 output_descriptor, d_value_conv, conv_descriptor,
                                 HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, model->workspace, model->workspace_size,
                                 &beta, input_descriptor, model->d_value_conv_weights);

    hipdnnConvolutionBackwardFilter(cudnn, &alpha, input_descriptor, batch_boards,
                                   output_descriptor, d_value_conv, conv_descriptor,
                                   HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0, model->workspace, model->workspace_size,
                                   &beta, model->value_conv_filter, model->d_value_conv_weights);

    // Combine gradients from policy and value heads
    float* d_res_output;
    hipMalloc(&d_res_output, model->config.batch_size * model->config.num_filters * model->config.input_height * model->config.input_width * sizeof(float));
    hipMemcpy(d_res_output, d_policy_conv, model->config.batch_size * model->config.num_filters * model->config.input_height * model->config.input_width * sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(d_res_output, d_value_conv, model->config.batch_size * model->config.num_filters * model->config.input_height * model->config.input_width * sizeof(float), hipMemcpyDeviceToDevice);

    // Residual blocks backward
    for (int i = model->config.num_residual_blocks - 1; i >= 0; i--) {
        float* d_res_input;
        hipMalloc(&d_res_input, model->config.batch_size * model->config.num_filters * model->config.input_height * model->config.input_width * sizeof(float));

        // Second convolution in residual block
        hipdnnConvolutionBackwardData(cudnn, &alpha, model->res_conv_filters[i*2+1], model->res_conv_weights[i*2+1],
                                     output_descriptor, d_res_output, conv_descriptor,
                                     HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, model->workspace, model->workspace_size,
                                     &beta, input_descriptor, d_res_input);

        hipdnnConvolutionBackwardFilter(cudnn, &alpha, input_descriptor, batch_boards,
                                       output_descriptor, d_res_output, conv_descriptor,
                                       HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0, model->workspace, model->workspace_size,
                                       &beta, model->res_conv_filters[i*2+1], model->d_res_conv_weights[i*2+1]);

        // First convolution in residual block
        hipdnnConvolutionBackwardData(cudnn, &alpha, model->res_conv_filters[i*2], model->res_conv_weights[i*2],
                                     output_descriptor, d_res_input, conv_descriptor,
                                     HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, model->workspace, model->workspace_size,
                                     &beta, input_descriptor, d_res_output);

        hipdnnConvolutionBackwardFilter(cudnn, &alpha, input_descriptor, batch_boards,
                                       output_descriptor, d_res_input, conv_descriptor,
                                       HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0, model->workspace, model->workspace_size,
                                       &beta, model->res_conv_filters[i*2], model->d_res_conv_weights[i*2]);

        // Add gradient from residual connection
        hipMemcpy(d_res_output, d_res_input, model->config.batch_size * model->config.num_filters * model->config.input_height * model->config.input_width * sizeof(float), hipMemcpyDeviceToDevice);

        hipFree(d_res_input);
    }

    // Input convolution backward
    hipdnnConvolutionBackwardData(cudnn, &alpha, model->input_conv_filter, model->input_conv_weights,
                                 output_descriptor, d_res_output, conv_descriptor,
                                 HIPDNN_CONVOLUTION_BWD_DATA_ALGO_0, model->workspace, model->workspace_size,
                                 &beta, input_descriptor, model->d_input_conv_weights);

    hipdnnConvolutionBackwardFilter(cudnn, &alpha, input_descriptor, batch_boards,
                                   output_descriptor, d_res_output, conv_descriptor,
                                   HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_0, model->workspace, model->workspace_size,
                                   &beta, model->input_conv_filter, model->d_input_conv_weights);

    // Cleanup
    hipFree(d_policy_conv);
    hipFree(d_value_conv);
    hipFree(d_res_output);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(conv_descriptor);
}

void adam_update(torch::optim::Adam& optimizer) {
    optimizer.step();
    optimizer.zero_grad();
}