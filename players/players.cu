#include "hip/hip_runtime.h"
#include "players.cuh"
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CHECK(call) { \
    hipError_t status = call; \
    if (status != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(status)); \
        exit(1); \
    } \
}

#define CURAND_CHECK(call) { \
    hiprandStatus_t status = call; \
    if (status != HIPRAND_STATUS_SUCCESS) { \
        fprintf(stderr, "CURAND error at %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(1); \
    } \
}

__device__ int random_action(const IGame* game, const int* board, int player, hiprandState* state) {
    int action_size = game->get_action_size(game);
    bool valid_moves[MAX_BOARD_SIZE];
    game->get_valid_moves_cuda(game, board, player, valid_moves);

    int valid_actions[MAX_BOARD_SIZE];
    int num_valid = 0;
    for (int i = 0; i < action_size; i++) {
        if (valid_moves[i]) {
            valid_actions[num_valid++] = i;
        }
    }

    if (num_valid == 0) return -1;  // No valid moves

    int random_index = hiprand(state) % num_valid;
    return valid_actions[random_index];
}

__global__ void random_action_kernel(const IGame* game, const int* board, int player, int* result) {
    hiprandState state;
    hiprand_init(clock64(), threadIdx.x, 0, &state);
    *result = random_action(game, board, player, &state);
}

int random_player_get_action(const IGame* game, const int* board, int player) {
    int* d_result;
    CUDA_CHECK(hipMalloc(&d_result, sizeof(int)));

    random_action_kernel<<<1, 1>>>(game, board, player, d_result);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    int result;
    CUDA_CHECK(hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost));
    
    CUDA_CHECK(hipFree(d_result));
    return result;
}

RandomPlayer* create_random_player() {
    RandomPlayer* player = (RandomPlayer*)malloc(sizeof(RandomPlayer));
    if (player == NULL) {
        fprintf(stderr, "Failed to allocate memory for RandomPlayer\n");
        exit(1);
    }
    player->base.get_action = random_player_get_action;
    return player;
}

int mcts_player_get_action(const IGame* game, const int* board, int player, MCTSState* mcts_state, float temperature) {
    mcts_update_with_move(mcts_state, -1);  // -1 to reset to root

    for (int i = 0; i < mcts_state->num_simulations; i++) {
        mcts_search(mcts_state);
    }

    return mcts_select_action(mcts_state, temperature);
}

MCTSPlayer* create_mcts_player(IGame* game, int num_simulations, float temperature) {
    MCTSPlayer* player = (MCTSPlayer*)malloc(sizeof(MCTSPlayer));
    if (player == NULL) {
        fprintf(stderr, "Failed to allocate memory for MCTSPlayer\n");
        exit(1);
    }
    player->base.get_action = (int (*)(const IGame*, const int*, int))mcts_player_get_action;
    player->mcts_state = mcts_init(game);
    player->mcts_state->num_simulations = num_simulations;
    player->temperature = temperature;
    return player;
}

int nnet_player_get_action(const IGame* game, const int* board, int player, INeuralNet* net, MCTSState* mcts_state, float temperature) {
    mcts_update_with_move(mcts_state, -1);  // -1 to reset to root

    for (int i = 0; i < mcts_state->num_simulations; i++) {
        mcts_search_with_nn(mcts_state, net);
    }

    return mcts_select_action(mcts_state, temperature);
}

NNetPlayer* create_nnet_player(IGame* game, INeuralNet* net, int num_simulations, float temperature) {
    NNetPlayer* player = (NNetPlayer*)malloc(sizeof(NNetPlayer));
    if (player == NULL) {
        fprintf(stderr, "Failed to allocate memory for NNetPlayer\n");
        exit(1);
    }
    player->base.get_action = (int (*)(const IGame*, const int*, int))nnet_player_get_action;
    player->net = net;
    player->mcts_state = mcts_init(game);
    player->mcts_state->num_simulations = num_simulations;
    player->temperature = temperature;
    return player;
}

void destroy_random_player(RandomPlayer* player) {
    free(player);
}

void destroy_mcts_player(MCTSPlayer* player) {
    mcts_free(player->mcts_state);
    free(player);
}

void destroy_nnet_player(NNetPlayer* player) {
    mcts_free(player->mcts_state);
    free(player);
}